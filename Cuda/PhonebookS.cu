#include "hip/hip_runtime.h"
//!pip install git+https://github.com/andreinechaev/nvcc4jupyter.git
//%load_ext nvcc_plugin
%%cu
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <hip/hip_runtime.h>
using namespace std;

struct Contact {
    char name[256];
    char phoneNumber[256];
};

__device__ bool deviceStrStr(const char* haystack, const char* needle, int needleLength) {
    for (int i = 0; i < needleLength; ++i) {
        if (haystack[i] != needle[i]) {
            return false;
        }
    }
    return true;
}

__global__ void searchContactsKernel(Contact* contacts, int size, const char* searchName, int searchNameLength) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size) {
        if (deviceStrStr(contacts[tid].name, searchName, searchNameLength)) {
            printf("Name: %s, Phone: %s\n", contacts[tid].name, contacts[tid].phoneNumber);
        }
    }
}

void printPhonebook(const std::vector<Contact>& phonebook) {
    std::cout << "Phonebook:\n";
    for (const auto& contact : phonebook) {
        std::cout << "Name: " << contact.name << ", Phone: " << contact.phoneNumber << "\n";
    }
    std::cout << "\n";
}

void searchContactsCUDA(const Contact* h_phonebook, int size, const char* searchName) {
    size_t contactsSize = size * sizeof(Contact);

    // Allocate device memory for contacts
    Contact* d_contacts;
    hipMalloc((void**)&d_contacts, contactsSize);

    // Copy contacts data from host to device
    hipMemcpy(d_contacts, h_phonebook, contactsSize, hipMemcpyHostToDevice);

    // Allocate device memory for searchName
    int searchNameLength = strlen(searchName) ;
    char* d_searchName;
    hipMalloc((void**)&d_searchName, searchNameLength);

    // Copy searchName to device
    hipMemcpy(d_searchName, searchName, searchNameLength, hipMemcpyHostToDevice);

    // Set up grid and block dimensions
    int blockSize = 256;
    int gridSize = (size + blockSize - 1) / blockSize;
    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    // Launch the search kernel
    searchContactsKernel<<<gridSize, blockSize>>>(d_contacts, size, d_searchName, searchNameLength);

    hipEventRecord(stop);
    // Synchronize to ensure kernel execution is complete
    hipEventSynchronize(stop);

    float milli = 0.0f;
    hipEventElapsedTime(&milli, start, stop);

    cout << "Time taken " << milli << endl;

    // Free device memory
    hipFree(d_contacts);
    hipFree(d_searchName);
}

void readPhonebook(const std::string& filename, std::vector<Contact> &phonebook) {
    std::ifstream file(filename);
    if (!file.is_open()) {
        std::cerr << "Error opening file: " << filename << std::endl;
       
    }

    Contact contact;
    while (file >> contact.name >> contact.phoneNumber) {
        phonebook.push_back(contact);
    }

    file.close();
    
}

int main() {
    const vector<string >phonebookFilenames = {"phonebook1.txt", "phonebook2.txt"};
    const char* searchName = "Jo";  // Change this to the desired search name

    // Read phonebook from file
    std::vector<Contact> phonebook ;
    for(auto filename:phonebookFilenames){

    readPhonebook(filename, phonebook);
    }

    if (!phonebook.empty()) {
        // Print the entire phonebook
       // printPhonebook(phonebook);

        // Search contacts using CUDA
        searchContactsCUDA(phonebook.data(), phonebook.size(), searchName);
    }

    return 0;
}